#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <cstdio>
#include <iostream>
#include <utility>
#include <random>
#include <omp.h>
#include "src/main.hxx"

using namespace std;




#define MAX_THREADS 64
#define MIN_COMPUTE_CUDA 10000000




template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1NormOmp(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}


template <class G>
void runPagerankBatch(const G& xo, int repeat, int steps, float batchFraction) {
  using T = float;
  using K = typename G::key_type;
  using E = typename G::edge_value_type;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  int span = int(1 * xo.span());
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  random_device dev;
  default_random_engine rnd(dev());

  for (int i=0; i<steps; i++) {
    auto x  = addSelfLoopsOmp(xo, E(), [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegreeOmp(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankMonolithicOmp(x, xt, init, {repeat});
    auto r0 = a0.ranks;

    // Add random edges for this batch.
    int batch = int(ceil(batchFraction * x.size()));
    auto yo = duplicate(xo);
    for (int i=0; i<int(ceil(0.2*batch)); i++)
      removeRandomEdge(yo, rnd);
    for (int i=0; i<int(ceil(0.8*batch)); i++)
      addRandomEdge(yo, rnd, span);
    // for (int i=0; i<batch; i++)
    //   addRandomEdgeByDegree(yo, rnd, span);
    updateOmpU(yo);
    auto y  = addSelfLoopsOmp(yo, E(), [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegreeOmp(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS + DELETIONS:
    // Adjust ranks for insertions + deletions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find Pagerank data.
    auto cs  = components(y, yt);
    auto b   = blockgraph(y, cs);
    auto bt  = transposeOmp(b);
    auto gs  = levelwiseGroupedComponentsFrom(cs, bt);
    auto [yks, yn] = dynamicVertices(x, xt, y, yt);
    auto [ycs, ym] = dynamicComponentIndices(x, xt, y, yt, cs, b);
    PagerankData<G> D {move(b), move(bt), move(cs)};
    printf("- batch update size: %d\n", batch);
    printf("- components: %d\n", b.order());
    printf("- blockgraph-levels: %zu\n", gs.size());
    printf("- affected-vertices: %zu\n", yn);
    printf("- affected-components: %zu\n", ym);

    // Find nvGraph-based pagerank.
    auto b0 = pagerankMonolithicOmp(y, yt, init, {repeat});
    // printRow(y, b0, b0, "pagerankNvgraph (static)");
    // auto c0 = pagerankNvgraph(y, yt, &s0, {repeat});
    // printRow(y, b0, c0, "pagerankNvgraph (incremental)");

    // Find OpenMP-based Monolithic pagerank.
    // auto b2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b2, "pagerankMonolithicOmp (static)");
    // auto c2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c2, "pagerankMonolithicOmp (incremental)");
    // auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d2, "pagerankMonolithicOmp (dynamic)");

    // Find OpenMP-based Monolithic pagerank (split).
    auto h2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li, 1, true}, &D);
    printRow(y, b0, h2, "pagerankMonolithicOmpSplit (static)");
    auto i2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, i2, "pagerankMonolithicOmpSplit (incremental)");
    auto j2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, j2, "pagerankMonolithicOmpSplit (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    // auto b3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, b3, "pagerankMonolithicCuda (static)");
    // auto c3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, c3, "pagerankMonolithicCuda (incremental)");
    // auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, d3, "pagerankMonolithicCuda (dynamic)");

    // Find CUDA-based Monolithic pagerank (split).
    // auto h3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, h3, "pagerankMonolithicCudaSplit (static)");
    // auto i3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, i3, "pagerankMonolithicCudaSplit (incremental)");
    // auto j3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, j3, "pagerankMonolithicCudaSplit (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b5, "pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c5, "pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d5, "pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    // auto b6 = pagerankLevelwiseCuda(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b6, "pagerankLevelwiseCuda (static)");
    // auto c6 = pagerankLevelwiseCuda(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c6, "pagerankLevelwiseCuda (incremental)");
    // auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d6, "pagerankLevelwiseCuda (dynamic)");
  }
}


template <class G>
void runPagerank(const G& x, int repeat) {
  vector<float> batches {1e-7, 1e-6, 1e-5, 1e-4, 1e-3, 1e-2, 1e-1};
  int M = x.size(), steps = 5;
  for (float batch : batches) {
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(x, repeat, steps, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  omp_set_num_threads(MAX_THREADS);
  printf("Loading graph %s ...\n", file);
  DiGraph<> x;
  readMtxOmpW(x, file); println(x);
  runPagerank(x, repeat);
  printf("\n");
  return 0;
}
