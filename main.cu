#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <cstdio>
#include <iostream>
#include <utility>
#include <random>
#include "src/main.hxx"

using namespace std;




#define MIN_COMPUTE_CUDA 10000000


struct GraphDelta {
  vector<pair<int, int>> deletions;
  vector<pair<int, int>> insertions;
};


template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}

template <class G>
void runPagerankBatch(const G& xr, const GraphDelta& delta, int repeat, int batch) {
  using T = float;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  int DD = delta.deletions.size();
  int DI = delta.insertions.size();
  auto xo = copy(xr);

  for (int di=0, ii=0; di<DD || ii<DI;) {
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankNvgraph(x, xt, init, {repeat});
    auto r0 = a0.ranks;

    // Add random edges for this batch.
    auto yo = copy(xo);
    int  db = di<=ii? min(ceilDiv(batch, 2), DD-di) : 0;
    int  ib = min(batch-db, DI-ii);
    for (int j=0; j<db; j++)
      removeEdge(yo, delta.deletions[di++]);
    for (int j=0; j<ib; j++)
      addEdge(yo, delta.insertions[ii++]);
    yo.correct();
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS + DELETIONS:
    // Adjust ranks for insertions + deletions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find Pagerank data.
    auto cs  = components(y, yt);
    auto b   = blockgraph(y, cs);
    auto bt  = transpose(b);
    auto gs  = levelwiseGroupedComponentsFrom(cs, bt);
    auto [yks, yn] = dynamicVertices(x, xt, y, yt);
    auto [ycs, ym] = dynamicComponentIndices(x, xt, y, yt, cs, b);
    PagerankData<G> D {move(b), move(bt), move(cs)};
    printf("- components: %d\n", b.order());
    printf("- blockgraph-levels: %d\n", gs.size());
    printf("- affected-vertices: %d\n", yn);
    printf("- affected-components: %d\n", ym);

    // Find nvGraph-based pagerank.
    auto b0 = pagerankNvgraph(y, yt, init, {repeat});
    printRow(y, b0, b0, "pagerankNvgraph (static)");
    auto c0 = pagerankNvgraph(y, yt, &s0, {repeat});
    printRow(y, b0, c0, "pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    // auto b1 = pagerankMonolithicSeq(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b1, "pagerankMonolithicSeq (static)");
    // auto c1 = pagerankMonolithicSeq(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c1, "pagerankMonolithicSeq (incremental)");
    // auto d1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d1, "pagerankMonolithicSeq (dynamic)");

    // Find sequential Monolithic pagerank (split).
    // auto h1 = pagerankMonolithicSeq(y, yt, init, {repeat, Li, 1, true}, &D);
    // printRow(y, b0, h1, "pagerankMonolithicSeqSplit (static)");
    // auto i1 = pagerankMonolithicSeq(y, yt, &s0, {repeat, Li, 1, true}, &D);
    // printRow(y, b0, i1, "pagerankMonolithicSeqSplit (incremental)");
    // auto j1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    // printRow(y, b0, j1, "pagerankMonolithicSeqSplit (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    // auto b2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b2, "pagerankMonolithicOmp (static)");
    // auto c2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c2, "pagerankMonolithicOmp (incremental)");
    // auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d2, "pagerankMonolithicOmp (dynamic)");

    // Find OpenMP-based Monolithic pagerank (split).
    auto h2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li, 1, true}, &D);
    printRow(y, b0, h2, "pagerankMonolithicOmpSplit (static)");
    auto i2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, i2, "pagerankMonolithicOmpSplit (incremental)");
    auto j2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, j2, "pagerankMonolithicOmpSplit (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    // auto b3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, b3, "pagerankMonolithicCuda (static)");
    // auto c3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, c3, "pagerankMonolithicCuda (incremental)");
    // auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, d3, "pagerankMonolithicCuda (dynamic)");

    // Find CUDA-based Monolithic pagerank (split).
    auto h3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, h3, "pagerankMonolithicCudaSplit (static)");
    auto i3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, i3, "pagerankMonolithicCudaSplit (incremental)");
    auto j3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, j3, "pagerankMonolithicCudaSplit (dynamic)");

    // Find sequential Levelwise pagerank.
    // auto b4 = pagerankLevelwiseSeq(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b4, "pagerankLevelwiseSeq (static)");
    // auto c4 = pagerankLevelwiseSeq(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c4, "pagerankLevelwiseSeq (incremental)");
    // auto d4 = pagerankLevelwiseSeqDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d4, "pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b5, "pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c5, "pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d5, "pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    auto b6 = pagerankLevelwiseCuda(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b6, "pagerankLevelwiseCuda (static)");
    auto c6 = pagerankLevelwiseCuda(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c6, "pagerankLevelwiseCuda (incremental)");
    auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d6, "pagerankLevelwiseCuda (dynamic)");

    // Move ahead.
    xo = move(yo);
  }
}


template <class G>
auto createMixedGraphDelta(const G& x, int del, int ins) {
  GraphDelta a;
  random_device dev;
  default_random_engine rnd(dev());
  for (int i=0; i<del; ++i)
    a.deletions.push_back(suggestRemoveRandomEdgeByDegree(x, rnd));
  for (int i=0; i<ins; ++i)
    a.insertions.push_back(suggestAddRandomEdgeByDegree(x, rnd, x.span()));
  return a;
}

template <class G>
void runPagerank(const G& x, int repeat) {
  vector<int> batches {1, 500, 1000, 2000};
  int steps = 1; int B = batches.back();
  for (int step=0; step<steps; ++step) {
    // printf("\n# Step %d\n", step);
    GraphDelta delta = createMixedGraphDelta(x, B/2, B/2);
    for (int batch : batches) {
      printf("\n# Batch size %.0e\n", (double) batch);
      runPagerankBatch(x, delta, repeat, batch);
    }
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Loading graph %s ...\n", file);
  auto x = readMtx(file); println(x);
  runPagerank(x, repeat);
  printf("\n");
  return 0;
}
