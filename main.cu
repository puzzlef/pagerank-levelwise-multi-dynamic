#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <cstdio>
#include <iostream>
#include <utility>
#include <random>
#include <omp.h>
#include "src/main.hxx"

using namespace std;




#define MAX_THREADS 64
#define MIN_COMPUTE_CUDA 10000000




template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}


template <class G>
void runPagerankBatch(const G& xo, int repeat, int steps, int batch) {
  using T = float;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  int span = int(1 * xo.span());
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  random_device dev;
  default_random_engine rnd(dev());

  for (int i=0; i<steps; i++) {
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankMonolithicOmp(x, xt, init, {repeat});
    auto r0 = a0.ranks;

    // Add random edges for this batch.
    auto yo = copy(xo);
    for (int i=0; i<batch/2; i++)
      removeRandomEdgeByDegree(yo, rnd);
    for (int i=0; i<ceilDiv(batch, 2); i++)
      addRandomEdgeByDegree(yo, rnd, span);
    // for (int i=0; i<batch; i++)
    //   addRandomEdgeByDegree(yo, rnd, span);
    yo.correct();
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS + DELETIONS:
    // Adjust ranks for insertions + deletions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find Pagerank data.
    auto cs  = components(y, yt);
    auto b   = blockgraph(y, cs);
    auto bt  = transpose(b);
    auto gs  = levelwiseGroupedComponentsFrom(cs, bt);
    auto [yks, yn] = dynamicVertices(x, xt, y, yt);
    auto [ycs, ym] = dynamicComponentIndices(x, xt, y, yt, cs, b);
    PagerankData<G> D {move(b), move(bt), move(cs)};
    printf("- components: %d\n", b.order());
    printf("- blockgraph-levels: %zu\n", gs.size());
    printf("- affected-vertices: %zu\n", yn);
    printf("- affected-components: %zu\n", ym);

    // Find nvGraph-based pagerank.
    auto b0 = pagerankMonolithicOmp(y, yt, init, {repeat});
    // printRow(y, b0, b0, "pagerankNvgraph (static)");
    // auto c0 = pagerankNvgraph(y, yt, &s0, {repeat});
    // printRow(y, b0, c0, "pagerankNvgraph (incremental)");

    // Find OpenMP-based Monolithic pagerank.
    // auto b2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b2, "pagerankMonolithicOmp (static)");
    // auto c2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c2, "pagerankMonolithicOmp (incremental)");
    // auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d2, "pagerankMonolithicOmp (dynamic)");

    // Find OpenMP-based Monolithic pagerank (split).
    auto h2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li, 1, true}, &D);
    printRow(y, b0, h2, "pagerankMonolithicOmpSplit (static)");
    auto i2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, i2, "pagerankMonolithicOmpSplit (incremental)");
    auto j2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, j2, "pagerankMonolithicOmpSplit (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    // auto b3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, b3, "pagerankMonolithicCuda (static)");
    // auto c3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, c3, "pagerankMonolithicCuda (incremental)");
    // auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, d3, "pagerankMonolithicCuda (dynamic)");

    // Find CUDA-based Monolithic pagerank (split).
    // auto h3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, h3, "pagerankMonolithicCudaSplit (static)");
    // auto i3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, i3, "pagerankMonolithicCudaSplit (incremental)");
    // auto j3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    // printRow(y, b0, j3, "pagerankMonolithicCudaSplit (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b5, "pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c5, "pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d5, "pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    // auto b6 = pagerankLevelwiseCuda(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b6, "pagerankLevelwiseCuda (static)");
    // auto c6 = pagerankLevelwiseCuda(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c6, "pagerankLevelwiseCuda (incremental)");
    // auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d6, "pagerankLevelwiseCuda (dynamic)");
  }
}


template <class G>
void runPagerank(const G& x, int repeat) {
  vector<int> batches {500, 1000, 2000, 5000, 10000};
  int M = x.size(), steps = 5;
  for (int batch : batches) {
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(x, repeat, steps, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  omp_set_num_threads(MAX_THREADS);
  printf("Loading graph %s ...\n", file);
  auto x = readMtx(file); println(x);
  runPagerank(x, repeat);
  printf("\n");
  return 0;
}
