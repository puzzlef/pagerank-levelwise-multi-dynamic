#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <cstdio>
#include <iostream>
#include <utility>
#include <random>
#include "src/main.hxx"

using namespace std;




#define MIN_COMPUTE_CUDA 10000000

template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}

template <class G>
void runPagerankBatch(const G& xo, int repeat, int steps, int batch) {
  using T = float;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  int span = int(1.1 * xo.span());
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  random_device dev;
  default_random_engine rnd(dev());

  for (int i=0; i<steps; i++) {
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankNvgraph(x, xt, init, {repeat});
    auto r0 = a0.ranks;

    // Add random edges for this batch.
    auto yo = copy(xo);
    for (int i=0; i<batch; i++)
      addRandomEdgeByDegree(yo, rnd, span);
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS:
    // Adjust ranks for insertions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find Pagerank data.
    auto cs  = components(y, yt);
    auto b   = blockgraph(y, cs);
    auto bt  = transpose(b);
    PagerankData<G> D {move(b), move(bt), move(cs)};

    // Find nvGraph-based pagerank.
    auto b0 = pagerankNvgraph(y, yt, init, {repeat});
    printRow(y, b0, b0, "I:pagerankNvgraph (static)");
    auto c0 = pagerankNvgraph(y, yt, &s0, {repeat});
    printRow(y, b0, c0, "I:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    // auto b1 = pagerankMonolithicSeq(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b1, "I:pagerankMonolithicSeq (static)");
    // auto c1 = pagerankMonolithicSeq(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c1, "I:pagerankMonolithicSeq (incremental)");
    // auto d1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d1, "I:pagerankMonolithicSeq (dynamic)");

    // Find sequential Monolithic pagerank (split).
    auto h1 = pagerankMonolithicSeq(y, yt, init, {repeat, Li, 1, true}, &D);
    printRow(y, b0, h1, "I:pagerankMonolithicSeqSplit (static)");
    auto i1 = pagerankMonolithicSeq(y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, i1, "I:pagerankMonolithicSeqSplit (incremental)");
    auto j1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, j1, "I:pagerankMonolithicSeqSplit (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    // auto b2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li}, &D);
    // printRow(y, b0, b2, "I:pagerankMonolithicOmp (static)");
    // auto c2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, c2, "I:pagerankMonolithicOmp (incremental)");
    // auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    // printRow(y, b0, d2, "I:pagerankMonolithicOmp (dynamic)");

    // Find OpenMP-based Monolithic pagerank (split).
    auto h2 = pagerankMonolithicOmp(y, yt, init, {repeat, Li, 1, true}, &D);
    printRow(y, b0, h2, "I:pagerankMonolithicOmpSplit (static)");
    auto i2 = pagerankMonolithicOmp(y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, i2, "I:pagerankMonolithicOmpSplit (incremental)");
    auto j2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, {repeat, Li, 1, true}, &D);
    printRow(y, b0, j2, "I:pagerankMonolithicOmpSplit (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    // auto b3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, b3, "I:pagerankMonolithicCuda (static)");
    // auto c3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, c3, "I:pagerankMonolithicCuda (incremental)");
    // auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA}, &D);
    // printRow(y, b0, d3, "I:pagerankMonolithicCuda (dynamic)");

    // Find CUDA-based Monolithic pagerank (split).
    auto h3 = pagerankMonolithicCuda(y, yt, init, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, h3, "I:pagerankMonolithicCudaSplit (static)");
    auto i3 = pagerankMonolithicCuda(y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, i3, "I:pagerankMonolithicCudaSplit (incremental)");
    auto j3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, {repeat, Li, MIN_COMPUTE_CUDA, true}, &D);
    printRow(y, b0, j3, "I:pagerankMonolithicCudaSplit (dynamic)");

    // Find sequential Levelwise pagerank.
    auto b4 = pagerankLevelwiseSeq(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b4, "I:pagerankLevelwiseSeq (static)");
    auto c4 = pagerankLevelwiseSeq(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c4, "I:pagerankLevelwiseSeq (incremental)");
    auto d4 = pagerankLevelwiseSeqDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d4, "I:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b5, "I:pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c5, "I:pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d5, "I:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    auto b6 = pagerankLevelwiseCuda(y, yt, init, {repeat, Li}, &D);
    printRow(y, b0, b6, "I:pagerankLevelwiseCuda (static)");
    auto c6 = pagerankLevelwiseCuda(y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, c6, "I:pagerankLevelwiseCuda (incremental)");
    auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, {repeat, Li}, &D);
    printRow(y, b0, d6, "I:pagerankLevelwiseCuda (dynamic)");

    // DELETIONS:
    // Adjust ranks for deletions.
    auto s1 = b0.ranks;
    vector<T> r1(x.span());
    adjustRanks(r1, s1, ks, ksOld, 0.0f, float(Y)/(X+1), 1.0f/(X+1));

    // Find Pagerank data.
    auto ds = components(x, xt);
    auto c  = blockgraph(x, ds);
    auto ct = transpose(c);
    PagerankData<G> E {move(c), move(ct), move(ds)};

    // Find nvGraph-based pagerank.
    auto e0 = pagerankNvgraph(x, xt, init, {repeat, Li});
    printRow(y, e0, e0, "D:pagerankNvgraph (static)");
    auto f0 = pagerankNvgraph(x, xt, &r1, {repeat, Li});
    printRow(y, e0, f0, "D:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    // auto e1 = pagerankMonolithicSeq(x, xt, init, {repeat, Li}, &E);
    // printRow(y, e0, e1, "D:pagerankMonolithicSeq (static)");
    // auto f1 = pagerankMonolithicSeq(x, xt, &r1, {repeat, Li}, &E);
    // printRow(y, e0, f1, "D:pagerankMonolithicSeq (incremental)");
    // auto g1 = pagerankMonolithicSeqDynamic(y, yt, x, xt, &r1, {repeat, Li}, &E);
    // printRow(y, e0, g1, "D:pagerankMonolithicSeq (dynamic)");

    // Find sequential Monolithic pagerank (split).
    auto k1 = pagerankMonolithicSeq(x, xt, init, {repeat, Li, 1, true}, &E);
    printRow(y, e0, k1, "D:pagerankMonolithicSeqSplit (static)");
    auto l1 = pagerankMonolithicSeq(x, xt, &r1, {repeat, Li, 1, true}, &E);
    printRow(y, e0, l1, "D:pagerankMonolithicSeqSplit (incremental)");
    auto m1 = pagerankMonolithicSeqDynamic(y, yt, x, xt, &r1, {repeat, Li, 1, true}, &E);
    printRow(y, e0, m1, "D:pagerankMonolithicSeqSplit (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    // auto e2 = pagerankMonolithicOmp(x, xt, init, {repeat, Li}, &E);
    // printRow(y, e0, e2, "D:pagerankMonolithicOmp (static)");
    // auto f2 = pagerankMonolithicOmp(x, xt, &r1, {repeat, Li}, &E);
    // printRow(y, e0, f2, "D:pagerankMonolithicOmp (incremental)");
    // auto g2 = pagerankMonolithicOmpDynamic(y, yt, x, xt, &r1, {repeat, Li}, &E);
    // printRow(y, e0, g2, "D:pagerankMonolithicOmp (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    auto k2 = pagerankMonolithicOmp(x, xt, init, {repeat, Li, 1, true}, &E);
    printRow(y, e0, k2, "D:pagerankMonolithicOmpSplit (static)");
    auto l2 = pagerankMonolithicOmp(x, xt, &r1, {repeat, Li, 1, true}, &E);
    printRow(y, e0, l2, "D:pagerankMonolithicOmpSplit (incremental)");
    auto m2 = pagerankMonolithicOmpDynamic(y, yt, x, xt, &r1, {repeat, Li, 1, true}, &E);
    printRow(y, e0, m2, "D:pagerankMonolithicOmpSplit (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    // auto e3 = pagerankMonolithicCuda(x, xt, init, {repeat, Li, MIN_COMPUTE_CUDA}, &E);
    // printRow(y, e0, e3, "D:pagerankMonolithicCuda (static)");
    // auto f3 = pagerankMonolithicCuda(x, xt, &r1, {repeat, Li, MIN_COMPUTE_CUDA}, &E);
    // printRow(y, e0, f3, "D:pagerankMonolithicCuda (incremental)");
    // auto g3 = pagerankMonolithicCudaDynamic(y, yt, x, xt, &r1, {repeat, Li, MIN_COMPUTE_CUDA}, &E);
    // printRow(y, e0, g3, "D:pagerankMonolithicCuda (dynamic)");

    // Find CUDA-based Monolithic pagerank (split).
    auto k3 = pagerankMonolithicCuda(x, xt, init, {repeat, Li, MIN_COMPUTE_CUDA, true}, &E);
    printRow(y, e0, k3, "D:pagerankMonolithicCudaSplit (static)");
    auto l3 = pagerankMonolithicCuda(x, xt, &r1, {repeat, Li, MIN_COMPUTE_CUDA, true}, &E);
    printRow(y, e0, l3, "D:pagerankMonolithicCudaSplit (incremental)");
    auto m3 = pagerankMonolithicCudaDynamic(y, yt, x, xt, &r1, {repeat, Li, MIN_COMPUTE_CUDA, true}, &E);
    printRow(y, e0, m3, "D:pagerankMonolithicCudaSplit (dynamic)");

    // Find sequential Levelwise pagerank.
    auto e4 = pagerankLevelwiseSeq(x, xt, init, {repeat, Li}, &E);
    printRow(y, e0, e4, "D:pagerankLevelwiseSeq (static)");
    auto f4 = pagerankLevelwiseSeq(x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, f4, "D:pagerankLevelwiseSeq (incremental)");
    auto g4 = pagerankLevelwiseSeqDynamic(y, yt, x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, g4, "D:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto e5 = pagerankLevelwiseOmp(x, xt, init, {repeat, Li}, &E);
    printRow(y, e0, e5, "D:pagerankLevelwiseOmp (static)");
    auto f5 = pagerankLevelwiseOmp(x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, f5, "D:pagerankLevelwiseOmp (incremental)");
    auto g5 = pagerankLevelwiseOmpDynamic(y, yt, x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, g5, "D:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    auto e6 = pagerankLevelwiseCuda(x, xt, init, {repeat, Li}, &E);
    printRow(y, e0, e6, "D:pagerankLevelwiseCuda (static)");
    auto f6 = pagerankLevelwiseCuda(x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, f6, "D:pagerankLevelwiseCuda (incremental)");
    auto g6 = pagerankLevelwiseCudaDynamic(y, yt, x, xt, &r1, {repeat, Li}, &E);
    printRow(y, e0, g6, "D:pagerankLevelwiseCuda (dynamic)");
  }
}


template <class G>
void runPagerank(const G& x, int repeat) {
  int M = x.size(), steps = 10;
  for (int batch=10, i=0; batch<M; batch*=i&1? 2:5, i++) {
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(x, repeat, steps, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Loading graph %s ...\n", file);
  auto x = readMtx(file); println(x);
  runPagerank(x, repeat);
  printf("\n");
  return 0;
}
